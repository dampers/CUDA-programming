
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32

__global__ void device_add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    c[index] = a[index] + b[index];
}

void fill_array(int *data)
{
    for(int idx=0;idx<N;idx++)
        data[idx] = idx;
}

void print_output(int *a, int *b, int *c)
{
    for(int idx=0;idx<N;idx++)
        printf("\n %d + %d = %d", a[idx], b[idx], c[idx]);
}

int main()
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int thread_per_block = 0, no_of_blocks = 0;
    int size = N * sizeof(int);
 
    a = (int *)malloc(size);
    fill_array(a);
    b = (int *)malloc(size);
    fill_array(b);
    c = (int *)malloc(size);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    thread_per_block = 4;
    no_of_blocks = N / thread_per_block;
    device_add<<<no_of_blocks,thread_per_block>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    print_output(a, b, c);
    free(a);free(b);free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
