
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VectorAdd(int *a, int *b, int *c)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a[tid] + b[tid];
}

int main()
{
	const int size = 512*65535;
	const int BufferSize = size*sizeof(int);

	int *InputA, *InputB, *Result;
	InputA = (int*)malloc(BufferSize);
	InputB = (int*)malloc(BufferSize);
	Result = (int*)malloc(BufferSize);
	
	int i = 0;
	for(i=0;i<size;i++)
	{
		InputA[i] = i;
		InputB[i] = i;
		Result[i] = 0;
	}

	int *dev_A, *dev_B, *dev_R;
	hipMalloc((void**)&dev_A, size*sizeof(int));
	hipMalloc((void**)&dev_B, size*sizeof(int));
	hipMalloc((void**)&dev_R, size*sizeof(int));

	hipMemcpy(dev_A, InputA, size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, InputB, size*sizeof(int), hipMemcpyHostToDevice);

	VectorAdd<<<65535, 512>>>(dev_A, dev_B, dev_R);
	hipMemcpy(Result, dev_R, size*sizeof(int), hipMemcpyDeviceToHost);

	for(i=0;i<5;i++)
		printf("Result[%d] : %d\n", i, Result[i]);
	printf("......\n");

	for(i=size-5;i<size;i++)
		printf("Result[%d] : %d\n", i, Result[i]);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_R);
	free(InputA);
	free(InputB);
	free(Result);

	return 0;
}